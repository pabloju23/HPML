#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__device__ double atomicAddDouble(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

// Definition of CUDA kernels
__global__ void calculatePi(double step, int num_steps, double* result) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double x;
    int window = num_steps / (gridDim.x * blockDim.x);
    int start = idx * window;
    int end = (idx + 1) * window;
    if (idx == (gridDim.x * blockDim.x) -1) end = num_steps;
    double sum = 0.0;
    for (int i = start; i < end; i ++) {
        x = (i + 0.5) * step;
        sum += 4.0 / (1.0 + x * x);
    }

    // Warp-level parallel reduction
    for (int offset = warpSize/2; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(0xffffffff, sum, offset);
    }

    // Use atomic operation to accumulate the results of each warp
    if (threadIdx.x % warpSize == 0) {
        atomicAddDouble(result, sum);
    }
}

int main(int argc, char* argv[]) {
    float t_seq, t_par, sp, ep;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Adjust the number of rectangles, threads, and blocks
    int num_steps = 100000;
    int num_threads = 256;
    int num_blocks = 1;

    if (argc == 4) {
        num_steps = atoi(argv[3]);
        num_threads = atoi(argv[1]);
        num_blocks = atoi(argv[2]);
        printf("Using %d threads, %d blocks and %d steps\n", num_threads, num_blocks, num_steps);
    } else if (argc != 1) {
        printf("Wrong number of parameters\n");
        printf("./a.out [ num_steps num_threads num_blocks ]\n");
        exit(-1);
    }

/*************************************/
/******** Computation of pi **********/
/*************************************/

  int i;
  double step = 1.0 / (double)num_steps;  
  double pi = 0.0;

  //
  // Sequential implementation
  //
  double x, sum = 0.0;
  hipEventRecord(start);  
  for (i=0; i<num_steps; i++){
     x = (i+0.5)*step;
     sum = sum + 4.0/(1.0+x*x);
  }
  pi = step * sum;
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  t_seq = 0;
  hipEventElapsedTime(&t_seq, start, stop);
  printf(" pi_seq = %20.15f\n", pi);
  printf(" time_seq = %20.15f\n", t_seq);

  //
  // Parallel implementation
  //
  
  // Defining the number of active threads
  int size = num_blocks * num_threads;

  // Call the CUDA
  // Allocate memory for the result on the device
  double* d_result;
  hipMalloc((void**)&d_result, sizeof(double));

  // Initialize result to 0
  hipMemset(d_result, 0, sizeof(double));
  hipEventRecord(start);
  // Call to the CUDA kernel
  calculatePi<<<num_blocks, num_threads>>>(step, num_steps, d_result);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  t_par = 0;
  hipEventElapsedTime(&t_par, start, stop);

  // Copy result from device to host
  hipMemcpy(&sum, d_result, sizeof(double), hipMemcpyDeviceToHost);

  // Free allocated memory
  hipFree(d_result);
  pi = step * sum;

  sp = t_seq / t_par;
  ep = sp / size;

  printf(" pi_par = %20.15f\n", pi);
  printf(" time_par = %20.15f, Sp = %20.15f , Ep = %20.15f\n", t_par, sp, ep);
}