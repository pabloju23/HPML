#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__device__ double atomicAddDouble(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

// CUDA kernel definition
__global__ void calculatePi(double step, int num_steps, double* result) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double x;
    int window = num_steps / (gridDim.x * blockDim.x);
    int start = idx * window;
    int end = (idx + 1) * window;
    if (idx == (gridDim.x * blockDim.x) -1) end = num_steps;
    double sum = 0.0;
    for (int i = start; i < end; i ++) {
        x = (i + 0.5) * step;
        sum += 4.0 / (1.0 + x * x);
    }
    atomicAddDouble(result, sum); // Use custom atomic operation to accumulate local result
}

int main(int argc, char* argv[]) {
    float t_seq, t_par, sp, ep;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Adjust the number of rectangles, threads, and blocks
    int num_steps = 100000;
    int num_threads = 256;
    int num_blocks = 1;

    if (argc == 4) {
        num_steps = atoi(argv[3]);
        num_threads = atoi(argv[1]);
        num_blocks = atoi(argv[2]);
        printf("Using %d threads, %d blocks and %d steps\n", num_threads, num_blocks, num_steps);
    } else if (argc != 1) {
        printf("Wrong number of parameters\n");
        printf("./a.out [ num_steps num_threads num_blocks ]\n");
        exit(-1);
    }

    /*************************************/
    /******** Computation of pi **********/
    /*************************************/

    int i;
    double step = 1.0 / (double)num_steps;
    double pi = 0.0;

    //
    // Sequential implementation
    //
    double x, sum = 0.0;
    hipEventRecord(start);
    for (i = 0; i < num_steps; i++) {
        x = (i + 0.5) * step;
        sum = sum + 4.0 / (1.0 + x * x);
    }
    pi = step * sum;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    t_seq = 0.0;
    hipEventElapsedTime(&t_seq, start, stop);
    
    printf(" pi_seq = %20.15f\n", pi);
    printf(" time_seq = %20.15f\n", t_seq);

    //
    // Parallel implementation
    //

    int size = num_blocks * num_threads;
    sum = 0.0;
    hipEventRecord(start);
    // Call the CUDA
    // Allocate memory for the result on the device
    double* d_result;
    hipMalloc((void**)&d_result, sizeof(double));

    // Initialize result to 0
    hipMemset(d_result, 0, sizeof(double));

    // Call the CUDA kernel
    calculatePi<<<num_blocks, num_threads>>>(step, num_steps, d_result);

    // Copy result from device to host
    hipMemcpy(&sum, d_result, sizeof(double), hipMemcpyDeviceToHost);
    // Free the memory
    hipFree(d_result);
    pi = step * sum;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    t_par = 0.0;
    hipEventElapsedTime(&t_par, start, stop);
    
    // Calculate speedup and efficiency
    sp = t_seq / t_par;
    ep = sp / size;

    printf(" pi_par = %20.15f\n", pi);
    printf(" time_par = %20.15f, Sp = %20.15f , Ep = %20.15f\n", t_par, sp, ep);
}